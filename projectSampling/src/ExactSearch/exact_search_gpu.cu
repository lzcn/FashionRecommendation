#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <errno.h>
// includes CUDA Runtime
#include "hip/hip_runtime.h"
#include ""
// find top 1K value
#define TOP_T 1024
#define Malloc(type,n) (type *)malloc((n)*sizeof(type))

const int threadsPerBlock = 4;
const int BlockSize = 500000;

static char *line = NULL;
static int max_line_len;

static char* readline(FILE *input)
{
	int len;
	
	if(fgets(line,max_line_len,input) == NULL)
		return NULL;

	while(strrchr(line,'\n') == NULL)
	{
		max_line_len *= 2;
		line = (char *) realloc(line,max_line_len);
		len = (int) strlen(line);
		if(fgets(line+len,max_line_len-len,input) == NULL)
			break;
	}
	return line;
}
//------------------------------------------------
// cuSubIndex is used for loop
/* 
	data: 
		index_cur: indicate the current index for 
		index_max: each element is the number vector
				of correspongding maxtirx
	method: 
		+ :add t to current index
*/
//------------------------------------------------
struct cuSubIndex
{
	int numMat;
	size_t *index_cur;
	const size_t *index_max;
	bool isDone;
	__device__ cuSubIndex(int n, size_t *max) :numMat(n){
		index_cur = (size_t*)malloc((n + 1)*sizeof(size_t));
		//index_max = (size_t*)malloc(n*sizeof(size_t));
		index_max = max;
		for (int i = 0; i < n + 1; ++i){
			//index_max[i] = max[i];
			index_cur[i] = 0;
		}
		//index_cur[numMat] = 0;
		isDone = false;
	}
	__device__ ~cuSubIndex(){
		//free(index_max);
		free(index_cur);
	}
	__device__ cuSubIndex&  operator+(const size_t step){
		size_t a, b;
		a = step;
		b = 0;
		size_t *tmp = (size_t *)malloc(numMat*sizeof(size_t));
		for (size_t i = 0; i < numMat; ++i){
			b = a % index_max[i];
			a = a / index_max[i];
			tmp[i] = b;
			index_cur[i] += b;
			while (index_cur[i] >= index_max[i]){
				index_cur[i] -= index_max[i];
				++index_cur[i + 1];
			}
		}
		if (a > 0){
			index_cur[numMat] += a;
		}
		if (index_cur[numMat] > 0){
			isDone = true;
		}
		free(tmp);
		return (*this);
	}
};


// data struct : matrix
typedef struct 
{
	size_t dim;
	size_t num;
	float *element;
}Matrix;

__host__ __device__ int doInsert(float value, float* toInsert, int num){
	float front,next;
	for(int i = 0; i < num; ++i){
		if(value > toInsert[i]){
			// find ans insert
			front = toInsert[i];
			toInsert[i] = value;
			// shift the left element
			for(int j = i + 1; j < num; ++j){
				next = toInsert[j];
				toInsert[j] = front;
				front = next;
			}
			return i;
		}

	}
	return num;
}
void mergeSort(float*src,float*dst, int *potIdxBlock);

__global__ void GetMaxValue(Matrix *dev_data, \
						int numMat, \
						size_t *dev_max_value, \
						int d, \
						float*max_value);

__global__ void getPot(float *src, int *potIdxBlock);
int main(int argc, char const *argv[])
{
	//----------------------------
	// check the input arguements
	//----------------------------
	if(argc <= 2){ 
		printf("Usage: exact_search <filename>\n");
		return -1;
	}
	char datafile[1024];
	char resultfile[1024];
	strcpy(datafile, argv[1]);
	strcpy(resultfile, argv[2]);
	printf("input file %s\n",datafile);
	FILE *fp;
	if((fp = fopen(datafile,"r") )== NULL){
		printf("file %s\"can not be find\"\n",datafile);
		return 1;
	}

	//------------------------------------------
	// file formate:
	// line 1: num0	num1 ...
	// line 2: dim
	// line 3: 0-th vector of 0-th vector set 
	//------------------------------------------

	//-----------------------------------
	// Load the numMat and dim from file
	//-----------------------------------
	printf(">> Loading data from file %s!\n", datafile);
	int numMat = 0;
	int dim = 0;
	max_line_len = 1024;
	line = (char *)malloc(max_line_len*sizeof(char));
	if(readline(fp) != NULL){
		char *p = strtok(line, " \t");
		++numMat;
		//printf("%s",*p);
		if(p == NULL || *p == '\n'){
			printf("Not NULL");
		}		
		while(1){
			p = strtok(NULL," \t");
			if(p == NULL || *p == '\n'){
				break;
			}
			++numMat;
		}
	}
	if(numMat == 0){
		fclose(fp);
		printf("Wrong formate or No data\n");
		return 0;
	}
	size_t *max_index = (size_t *)malloc(numMat*sizeof(size_t));
	rewind(fp);
	if(readline(fp) != NULL){
		char *p = strtok(line, " \t");
		max_index[0] = atoi(p);
		for(int i = 1; i < numMat; ++i){
			char *p = strtok(NULL," \t");
			max_index[i] = atoi(p);
		}
	}
	if(readline(fp) != NULL){
		dim = atoi(line);
	}
	//-----------------------------------
	// Load data's element from file
	//-----------------------------------
	Matrix *data = (Matrix*)malloc(numMat*sizeof(Matrix));
	for(int i = 0; i < numMat; ++i){
		data[i].num = max_index[i];
		data[i].dim = dim;
		data[i].element = (float*)malloc(max_index[i]*dim*sizeof(float));
		for(size_t j = 0; j < max_index[i]; ++j){
			char *p;
			if(readline(fp) != NULL){
				p = strtok(line," \t");
				data[i].element[j*dim] = strtod(p,NULL);
			}
			for(size_t k = 1; k < dim; ++k){
				p = strtok(NULL,"\t");
				data[i].element[j*dim + k] = strtod(p, NULL);
			}
		}
	}
	fclose(fp);
	//-----------------------------------
	// Copy the data in host to device
	//-----------------------------------

	Matrix *h_data = (Matrix*)malloc(numMat*sizeof(Matrix));
	memcpy(h_data, data, numMat *sizeof(Matrix));
	// max index stored in device
	size_t *dev_index_max;
	hipMalloc(&dev_index_max,numMat*sizeof(size_t));
	hipMemcpy(dev_index_max, max_index, numMat*sizeof(size_t), hipMemcpyHostToDevice);
	for(int i = 0; i < numMat; ++i){
		hipMalloc( &(h_data[i].element), \
					data[i].dim*data[i].num*sizeof(float));
		hipMemcpy( h_data[i].element, \
					data[i].element, \
					data[i].dim*data[i].num*sizeof(float), \
					hipMemcpyHostToDevice);
	}

	Matrix* dev_data;
	hipMalloc(&dev_data, numMat*sizeof(Matrix));
	hipMemcpy(dev_data, h_data, numMat*sizeof(Matrix), hipMemcpyHostToDevice);
	//----------------------------------------------------
	// Varible to save the top BlockSize parts of 1K value
	//----------------------------------------------------
	float *max_value,*dev_max_value;
	hipMalloc(&dev_max_value, BlockSize*TOP_T*sizeof(float));
	max_value = (float*)malloc(BlockSize*TOP_T*sizeof(float));
	printf(">> Laoding data comleted!\n");
	//-----------------
	// Invoke kernel
	//-----------------
	printf(">> Starting exact search!\n");
	hipEvent_t start, stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	GetMaxValue<<<BlockSize,threadsPerBlock>>>(dev_data, \
											   numMat, \
											   dev_index_max, \
											   dim, \
											   dev_max_value);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf(">> GetMaxValue costs %f\n", elapsedTime);
	hipDeviceSynchronize();
	//-------------------------------------------------
	// merge the BlockSize of TOP_T values into one
	//-------------------------------------------------
	int *potIdxBlock;
	hipMalloc(&potIdxBlock, BlockSize*sizeof(int));
	hipEventRecord(&start, 0);
	getPot<<<BlockSize,1>>>(dev_max_value,potIdxBlock);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf(">> getPot costs %f\n", elapsedTime);
	
	//--------------------------------------------------
	// Copy result from device memory to host memory
	//--------------------------------------------------
	int *h_potIdxBlock;
	h_potIdxBlock = (int*)malloc(BlockSize*sizeof(int));
	hipMemcpy(h_potIdxBlock,potIdxBlock,BlockSize*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(max_value,dev_max_value,BlockSize*TOP_T*sizeof(float),hipMemcpyDeviceToHost);
	//------------------------------------
	// Get the top 1K of max_value
	//------------------------------------
	float *top_t = (float*)malloc(TOP_T*sizeof(float));
	mergeSort(max_value,top_t,h_potIdxBlock);
	/*
	for(int i = 1; i < BlockSize; ++i){
		for(int j = 0; j < TOP_T; ++j){
			if(max_value[i*TOP_T + j] > max_value[TOP_T-1]){
				//insert
				doInsert(max_value[i*TOP_T + j], max_value, TOP_T);
			}
		}
	}
	*/
	//----------------------------
	// write the top-1K to file
	//----------------------------
	FILE *fp_out = fopen(resultfile, "w");
	for (int i = 0; i < TOP_T; ++i){
		fprintf(fp_out,"%f\n",top_t[i]);
		//fprintf(fp_out,"%f\n",max_value[i]);
	}
	fclose(fp_out);
	//-----------------------------------
	// Free device memory and host memory
	//-----------------------------------
	for (int i = 0; i < numMat; ++i){
		hipFree(h_data[i].element);
		free(data[i].element); 
	}
	hipFree(dev_index_max);
	hipFree(dev_data);
	hipFree(dev_max_value);
	hipFree(potIdxBlock);
	free(line);
	free(max_index);
	free(data);
	free(h_data);
	free(max_value);
	free(h_potIdxBlock);
	free(top_t);

	return 0;
}

__global__ void GetMaxValue(Matrix *dev_data,\
							int numMat, \
							size_t *max_index, \
							int d, \
							float *max_value){
	// shared memory to save threadsPerBlock parts of top 1K value
	__shared__ float cache[threadsPerBlock][TOP_T];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;


	float *thread_max_value;
	float *mul_value;
	float tmp_max;
	struct cuSubIndex subIndex(numMat, max_index);
	thread_max_value = (float*)malloc(TOP_T*sizeof(float));
	mul_value = (float*)malloc(d*sizeof(float));

	for(int i = 0; i < TOP_T; ++i){
		thread_max_value[i] = 0;
	}
	for(int i = 0; i < d; ++i){
		mul_value[i] = 1;
	}

	subIndex = subIndex + tid;
	while(!(subIndex.isDone)){
		for(int i = 0; i < numMat; ++i){
			for(int j = 0; j < d; ++j){
				mul_value[j] *= dev_data[i].element[d*(subIndex.index_cur[i]) + j];
			}
		}
		for(int i = 0; i < d; ++i){
			tmp_max += mul_value[i];
		}
		if(tmp_max > thread_max_value[TOP_T - 1]){
			doInsert(tmp_max,thread_max_value,TOP_T);
		}
		subIndex = subIndex + blockDim.x * gridDim.x;
	}
	// put the top 1K value of thread to global memory
	for(int i = 0; i < TOP_T; ++i){
		cache[threadIdx.x][i] = thread_max_value[i];
	}
	__syncthreads();

	int cacheIndex = blockDim.x/2;
	// merge two 1K value vector into one vector
	while(cacheIndex != 0){
		if(threadIdx.x < cacheIndex){
			for(int i = 0; i < TOP_T; ++i){
				if(cache[threadIdx.x + cacheIndex][i] > cache[threadIdx.x][TOP_T - 1]){
					doInsert(cache[threadIdx.x + cacheIndex][i],\
							&(cache[threadIdx.x][0]),TOP_T);
				}
			}
		}
		__syncthreads();
		cacheIndex /= 2;
	}
	// cache[0][:] is the final top 1K value
	// put it into global memory.
	// final BlockSize 1K value vector
	if(threadIdx.x == 0){
		for(int i = 0; i < TOP_T; ++i){
			max_value[TOP_T*blockIdx.x + i] = cache[0][i];
		}
	}
	free(thread_max_value);
	free(mul_value);
}


__global__ void getPot(float *src, int *potIdxBlock){
	double temp;
	int index = 0;
	bool doneSearch = false;
	size_t pos = blockIdx.x*TOP_T;
	float *potV = (float*)malloc(BlockSize*sizeof(float));
	for(int i = 0; i < TOP_T; ++i){
		// the order of cache[cache_pos + i] is at least i
		temp = src[pos + i];
		index = i;
		doneSearch = false;
		// compare this value to others
		// if encounter a value is other beam which bigger than it
		// increment index and go on, otherwise 
		// encountering a value no bigger than it
		// go to another beam to do comparison
		for(int m = 0; m < blockDim.x; ++m){
			// escape self beam
			if(m == blockIdx.x) continue; 
			// search in other beam
			for(int k = 0; k < TOP_T; ++k){
				// encounter a bigger value 
				if(temp < src[pos + k]){
					++index;
				}else{
					break;
				}
				// if this value is out of the top-t
				// we can finish searching
				if(index >= TOP_T){
					doneSearch = true;
					break;
				}
			}
		}
		// if beam[i] has no potential to be the top-t
		// then the number of elements have the potential
		// is i then record it;
		if(doneSearch){
			potIdxBlock[blockIdx.x] = i;
			break;
		}
  	}
  	
}

void mergeSort(float*src,float*dst, int *potIdxBlock){
  	int count = 0;
 	for(int m = 0; m < BlockSize; ++m){
		count += potIdxBlock[m];
	}
	float *potV = (float*)malloc(count*sizeof(float));
	int p = 0;
 	for(int m = 0; m < BlockSize; ++m){
		for(int i = 0 ; i < potIdxBlock[m]; ++i){
			potV[p] = src[m*TOP_T + i];
			++p;
		}
	}
	//sort potV
	for(int i = 1; i < count; i++){  
		if(potV[i] > potV[i-1]){
		    int j = i - 1;
		    float x = potV[i];
		    potV[i] = potV[i-1];
		    while(j >= 0 && x > potV[j]){
		        potV[j + 1] = potV[j];
		        j--;
		    }
		    potV[j+1] = x;
		}  
	}
	for(int i = 0; i < TOP_T; ++i){
		dst[i] = potV[i];
	}
	free(potV);
}