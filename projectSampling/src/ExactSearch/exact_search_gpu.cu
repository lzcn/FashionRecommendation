#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <errno.h>
// includes CUDA Runtime
#include "hip/hip_runtime.h"

// find top 1K value
#define TOP1K 1000
#define Malloc(type,n) (type *)malloc((n)*sizeof(type))

const int threadsPerBlock = 4;
const int BlockSize = 2000;

static char *line = NULL;
static int max_line_len;

static char* readline(FILE *input)
{
	int len;
	
	if(fgets(line,max_line_len,input) == NULL)
		return NULL;

	while(strrchr(line,'\n') == NULL)
	{
		max_line_len *= 2;
		line = (char *) realloc(line,max_line_len);
		len = (int) strlen(line);
		if(fgets(line+len,max_line_len-len,input) == NULL)
			break;
	}
	return line;
}
//------------------------------------------------
// cuSubIndex is used for loop
/* 
	data: 
		index_cur: indicate the current index for 
		index_max: each element is the number vector
				of correspongding maxtirx
	method: 
		+ :add t to current index
*/
//------------------------------------------------
struct cuSubIndex
{
	int numMat;
	size_t *index_cur;
	const size_t *index_max;
	bool isDone;
	__device__ cuSubIndex(int n, size_t *max) :numMat(n){
		index_cur = (size_t*)malloc((n + 1)*sizeof(size_t));
		//index_max = (size_t*)malloc(n*sizeof(size_t));
		index_max = max;
		for (int i = 0; i < n + 1; ++i){
			//index_max[i] = max[i];
			index_cur[i] = 0;
		}
		//index_cur[numMat] = 0;
		isDone = false;
	}
	__device__ ~cuSubIndex(){
		//free(index_max);
		free(index_cur);
	}
	__device__ cuSubIndex&  operator+(const size_t step){
		size_t a, b;
		a = step;
		b = 0;
		size_t *tmp = (size_t *)malloc(numMat*sizeof(size_t));
		for (size_t i = 0; i < numMat; ++i){
			b = a % index_max[i];
			a = a / index_max[i];
			tmp[i] = b;
			index_cur[i] += b;
			while (index_cur[i] >= index_max[i]){
				index_cur[i] -= index_max[i];
				++index_cur[i + 1];
			}
		}
		if (a > 0){
			index_cur[numMat] += a;
		}
		if (index_cur[numMat] > 0){
			isDone = true;
		}
		free(tmp);
		return (*this);
	}
};


// data struct : matrix
typedef struct 
{
	size_t dim;
	size_t num;
	float *element;
}Matrix;

__host__ __device__ int doInsert(float value, float* toInsert, int num){
	float front,next;
	for(int i = 0; i < num; ++i){
		if(value > toInsert[i]){
			// find ans insert
			front = toInsert[i];
			toInsert[i] = value;
			// shift the left element
			for(int j = i + 1; j < num; ++j){
				next = toInsert[j];
				toInsert[j] = front;
				front = next;
			}
			return i;
		}

	}
	return num;
}

__global__ void GetMaxValue(Matrix *dev_data, \
						int numMat, \
						size_t *dev_max_value, \
						int d, \
						float*max_value);

int main(int argc, char const *argv[])
{
	//----------------------------
	// check the input arguements
	//----------------------------
	if(argc <= 2){ 
		printf("Usage: exact_search <filename>\n");
		return -1;
	}
	char datafile[1024];
	char resultfile[1024];
	strcpy(datafile, argv[1]);
	strcpy(resultfile, argv[2]);
	printf("input file %s\n",datafile);
	FILE *fp;
	if((fp = fopen(datafile,"r") )== NULL){
		printf("file %s\"can not be find\"\n",datafile);
		return 1;
	}

	//------------------------------------------
	// file formate:
	// line 1: num0	num1 ...
	// line 2: dim
	// line 3: 0-th vector of 0-th vector set 
	//------------------------------------------

	//-----------------------------------
	// Load the numMat and dim from file
	//-----------------------------------
	printf(">> Loading data from file %s!\n", datafile);
	int numMat = 0;
	int dim = 0;
	max_line_len = 1024;
	line = (char *)malloc(max_line_len*sizeof(char));
	if(readline(fp) != NULL){
		char *p = strtok(line, " \t");
		++numMat;
		//printf("%s",*p);
		if(p == NULL || *p == '\n'){
			printf("Not NULL");
		}		
		while(1){
			p = strtok(NULL," \t");
			if(p == NULL || *p == '\n'){
				break;
			}
			++numMat;
		}
	}
	if(numMat == 0){
		fclose(fp);
		printf("Wrong formate or No data\n");
		return 0;
	}
	size_t *max_index = (size_t *)malloc(numMat*sizeof(size_t));
	rewind(fp);
	if(readline(fp) != NULL){
		char *p = strtok(line, " \t");
		max_index[0] = atoi(p);
		for(int i = 1; i < numMat; ++i){
			char *p = strtok(NULL," \t");
			max_index[i] = atoi(p);
		}
	}
	if(readline(fp) != NULL){
		dim = atoi(line);
	}
	printf(">> Laoding data comleted!\n");
	//-----------------------------------
	// Load data's element from file
	//-----------------------------------
	printf(">> Starting exact search!\n");
	Matrix *data = (Matrix*)malloc(numMat*sizeof(Matrix));
	for(int i = 0; i < numMat; ++i){
		data[i].num = max_index[i];
		data[i].dim = dim;
		data[i].element = (float*)malloc(max_index[i]*dim*sizeof(float));
		for(size_t j = 0; j < max_index[i]; ++j){
			char *p;
			if(readline(fp) != NULL){
				p = strtok(line," \t");
				data[i].element[j*dim] = strtod(p,NULL);
			}
			for(size_t k = 1; k < dim; ++k){
				p = strtok(NULL,"\t");
				data[i].element[j*dim + k] = strtod(p, NULL);
			}
		}
	}
	fclose(fp);
	//-----------------------------------
	// Copy the data in host to device
	//-----------------------------------

	Matrix *h_data = (Matrix*)malloc(numMat*sizeof(Matrix));
	memcpy(h_data, data, numMat *sizeof(Matrix));
	// max index stored in device
	size_t *dev_index_max;
	hipMalloc(&dev_index_max,numMat*sizeof(size_t));
	hipMemcpy(dev_index_max, max_index, numMat*sizeof(size_t), hipMemcpyHostToDevice);
	for(int i = 0; i < numMat; ++i){
		hipMalloc( &(h_data[i].element), \
					data[i].dim*data[i].num*sizeof(float));
		hipMemcpy( h_data[i].element, \
					data[i].element, \
					data[i].dim*data[i].num*sizeof(float), \
					hipMemcpyHostToDevice);
	}

	Matrix* dev_data;
	hipMalloc(&dev_data, numMat*sizeof(Matrix));
	hipMemcpy(dev_data, h_data, numMat*sizeof(Matrix), hipMemcpyHostToDevice);
	//----------------------------------------------------
	// Varible to save the top BlockSize parts of 1K value
	//----------------------------------------------------
	float *max_value,*dev_max_value;
	hipMalloc(&dev_max_value, BlockSize*TOP1K*sizeof(float));
	max_value = (float*)malloc(BlockSize*TOP1K*sizeof(float));
	//-----------------
	// Invoke kernel
	//-----------------
	GetMaxValue<<<BlockSize,threadsPerBlock>>>(dev_data, \
											   numMat, \
											   dev_index_max, \
											   dim, \
											   dev_max_value);
	//--------------------------------------------------
	// Copy result from device memory to host memory
	//--------------------------------------------------
	hipMemcpy(max_value,dev_max_value,BlockSize*TOP1K*sizeof(float),hipMemcpyDeviceToHost);
	//------------------------------------
	// Get the top 1K of max_value
	//------------------------------------
	for(int i = 1; i < BlockSize; ++i){
		for(int j = 0; j < TOP1K; ++j){
			if(max_value[i*TOP1K + j] > max_value[TOP1K-1]){
				//insert
				doInsert(max_value[i*TOP1K + j], max_value, TOP1K);
			}
		}
	}
	//----------------------------
	// write the top-1K to file
	//----------------------------
	FILE *fp_out = fopen(resultfile, "w");
	for (int i = 0; i < TOP1K; ++i){
		fprintf(fp_out,"%f\n",max_value[i]);
	}
	fclose(fp_out);
	//-----------------------------------
	// Free device memory and host memory
	//-----------------------------------
	for (int i = 0; i < numMat; ++i){
		hipFree(h_data[i].element);
		free(data[i].element); 
	}
	hipFree(dev_index_max);
	hipFree(dev_data);
	hipFree(dev_max_value);
	free(line);
	free(max_index);
	free(data);
	free(h_data);
	free(max_value);

	return 0;
}

__global__ void GetMaxValue(Matrix *dev_data,\
							int numMat, \
							size_t *max_index, \
							int d, \
							float *max_value){
	// shared memory to save threadsPerBlock parts of top 1K value
	__shared__ float cache[threadsPerBlock][TOP1K];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;


	float *thread_max_value;
	float *mul_value;
	float tmp_max;
	struct cuSubIndex subIndex(numMat, max_index);
	thread_max_value = (float*)malloc(TOP1K*sizeof(float));
	mul_value = (float*)malloc(d*sizeof(float));

	for(int i = 0; i < TOP1K; ++i){
		thread_max_value[i] = 0;
	}
	for(int i = 0; i < d; ++i){
		mul_value[i] = 1;
	}

	subIndex = subIndex + tid;
	while(!(subIndex.isDone)){
		for(int i = 0; i < numMat; ++i){
			for(int j = 0; j < d; ++j){
				mul_value[j] *= dev_data[i].element[d*(subIndex.index_cur[i]) + j];
			}
		}
		for(int i = 0; i < d; ++i){
			tmp_max += mul_value[i];
		}
		if(tmp_max > thread_max_value[TOP1K - 1]){
			doInsert(tmp_max,thread_max_value,TOP1K);
		}
		subIndex = subIndex + blockDim.x * gridDim.x;
	}
	// put the top 1K value of thread to global memory
	for(int i = 0; i < TOP1K; ++i){
		cache[threadIdx.x][i] = thread_max_value[i];
	}
	__syncthreads();

	int cacheIndex = blockDim.x/2;
	// merge two 1K value vector into one vector
	while(cacheIndex != 0){
		if(threadIdx.x < cacheIndex){
			for(int i = 0; i < TOP1K; ++i){
				if(cache[threadIdx.x + cacheIndex][i] > cache[threadIdx.x][TOP1K - 1]){
					doInsert(cache[threadIdx.x + cacheIndex][i],\
							&(cache[threadIdx.x][0]),TOP1K);
				}
			}
		}
		__syncthreads();
		cacheIndex /= 2;
	}
	// cache[0][:] is the final top 1K value
	// put it into global memory.
	// final BlockSize 1K value vector
	if(threadIdx.x == 0){
		for(int i = 0; i < TOP1K; ++i){
			max_value[TOP1K*blockIdx.x + i] = cache[0][i];
		}
	}
	free(thread_max_value);
	free(mul_value);
}
