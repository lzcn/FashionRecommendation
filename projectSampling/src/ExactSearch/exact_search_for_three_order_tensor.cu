#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <errno.h>
// includes CUDA Runtime
#include "hip/hip_runtime.h"

// find top 1024 value
#define top_t 1024

const dim3 blockSize(16, 16, 1);
const dim3 gridSize(64, 64, 1);
const size_t BlockSize = gridSize.x * gridSize.y;

static char *line = NULL;
static int max_line_len;

static char* readline(FILE *input)
{
	int len;
	
	if(fgets(line,max_line_len,input) == NULL)
		return NULL;

	while(strrchr(line,'\n') == NULL)
	{
		max_line_len *= 2;
		line = (char *) realloc(line,max_line_len);
		len = (int) strlen(line);
		if(fgets(line+len,max_line_len-len,input) == NULL)
			break;
	}
	return line;
}

/* 
	cuSubIndex is used for loop
	data: 
		x, y: indicate the current index;
		xLoop, yLoop: the max for x and y;
*/
struct cuSubIndex
{
	size_t x;
	size_t y;
	size_t xLoop;
	size_t yLoop;
	bool isDone;
	__device__ cuSubIndex(size_t a,size_t b){
		x = 0;
		y = 0;
		xLoop = a;
		yLoop = b;
		isDone = false;
	}
	__device__ ~cuSubIndex(){}
	__device__ cuSubIndex& operator++(){
    	++x;
    	if(x == xLoop){
    		x = 0;
    		++y;
    	}
    	if(y == yLoop){
    		isDone = true;
    	}
    	return (*this);
	}
};


/* 
	matrix
*/
typedef struct 
{
	// dimension of each vector
	size_t dim;
	// number of vectors
	size_t num;
	// element of the matrix
	float *element;
}Matrix;

/*
	insert the value to a sorted list
	value: the value to be inserted
	toInset: the list
	num: the length of list
*/
__device__ int doInsert(float value, float* toInsert, int num){
	float front,next;
	for(int i = 0; i < num; ++i){
		// find where to insert
		if(value > toInsert[i]){
			// insert the value before i
			front = toInsert[i];
			toInsert[i] = value;
			// shift the left element
			for(int j = i + 1; j < num; ++j){
				next = toInsert[j];
				toInsert[j] = front;
				front = next;
			}
			// return the insert position
			return i;
		}

	}
	return num;
}

/*
	h_maxValue has size gridSize*top-t;
	each block will compute their own top-t value and 
	then save into the corresponding position in h_maxValue;
	numXaxis, numYaxis : the number of vector of 
	Matrix X and Matrix Y;
*/
__global__ void GetMaxValue(Matrix *dev_data,\
							size_t numXaxis, \
							size_t numYaxis, \
							int dim, \
							float *h_maxValue);
/*
	see h_maxValue has gridSize vectors
	for each vector containing top-t values in h_maxValue
	find how many values the vector has that has the potential 
	to be in order top-t in global;
*/
__global__ void getPot(float*h_maxValue,int*potIdxBlock);

/*
	merge the h_maxValue into one vector (top-t)
	according to the potential we get;
*/
__global__ void mergeSort(float*h_maxValue,float*dst,int*potIdxBlock);

/*	
	main function
	input: data-filename, out-filename
*/
int main(int argc, char const *argv[])
{
	//----------------------------
	// check the input arguments
	//----------------------------
	if(argc <= 2){ 
		printf("Usage: exact_search <filename>\n");
		return -1;
	}
	char datafile[1024];
	char resultfile[1024];
	strcpy(datafile, argv[1]);
	strcpy(resultfile, argv[2]);
	printf("input file %s\n",datafile);
	FILE *fp;
	if((fp = fopen(datafile,"r") )== NULL){
		printf("file %s\"can not be find\"\n",datafile);
		return 1;
	}

	//------------------------------------------
	// file formate:
	// line 1: num_0 num_1 num_2
	// line 2: dim
	// line 3: 0-th vector of 0-th vector set 
	//------------------------------------------

	//-----------------------------------
	// Load the numMat and dim from file
	//-----------------------------------
	printf(">> Loading data from file %s!\n", datafile);
	int numMat = 0;
	int dim = 0;
	max_line_len = 1024;
	line = (char *)malloc(max_line_len*sizeof(char));
	if(readline(fp) != NULL){
		char *p = strtok(line, " \t");
		++numMat;
		if(p == NULL || *p == '\n'){
			printf("Not NULL");
		}		
		while(1){
			p = strtok(NULL," \t");
			if(p == NULL || *p == '\n'){
				break;
			}
			++numMat;
		}
	}
	if(numMat != 3){
		fclose(fp);
		printf("Wrong formate or No data\n");
		return 0;
	}
	size_t *max_index = (size_t *)malloc(numMat*sizeof(size_t));
	rewind(fp);
	if(readline(fp) != NULL){
		char *p = strtok(line, " \t");
		max_index[0] = atoi(p);
		for(int i = 1; i < numMat; ++i){
			char *p = strtok(NULL," \t");
			max_index[i] = atoi(p);
		}
	}
	if(readline(fp) != NULL){
		dim = atoi(line);
	}
	//-----------------------------------
	// Load data's element from file
	//-----------------------------------
	Matrix *data = (Matrix*)malloc(numMat*sizeof(Matrix));
	for(int i = 0; i < numMat; ++i){
		data[i].num = max_index[i];
		data[i].dim = dim;
		data[i].element = (float*)malloc(max_index[i]*dim*sizeof(float));
		for(size_t j = 0; j < max_index[i]; ++j){
			char *p;
			if(readline(fp) != NULL){
				p = strtok(line," \t");
				data[i].element[j*dim] = strtod(p,NULL);
			}
			for(size_t k = 1; k < dim; ++k){
				p = strtok(NULL,"\t");
				data[i].element[j*dim + k] = strtod(p, NULL);
			}
		}
	}
	fclose(fp);
	//-----------------------------------
	// Copy the data in host to device
	//-----------------------------------
	printf(">> Copying data to device!\n");
	Matrix *h_data = (Matrix*)malloc(numMat*sizeof(Matrix));
	memcpy(h_data, data, numMat *sizeof(Matrix));
	for(int i = 0; i < numMat; ++i){
		hipMalloc( &(h_data[i].element), \
					data[i].dim*data[i].num*sizeof(float));
		hipMemcpy( h_data[i].element, \
					data[i].element, \
					data[i].dim*data[i].num*sizeof(float), \
					hipMemcpyHostToDevice);
	}
	Matrix* dev_data;
	hipMalloc(&dev_data, numMat*sizeof(Matrix));
	hipMemcpy(dev_data, h_data, numMat*sizeof(Matrix), hipMemcpyHostToDevice);
	
	printf(">> Loading data completed!\n");
	//--------------------------------------------------------------
	// Create variables to save the top 1024 value of each block
	//--------------------------------------------------------------
	printf(">> Starting search!\n");
	float *h_maxValue,*d_maxValue;
	float *h_top_t, *d_top_t;
	// d_maxValue is the place to save the top-t values of each block
	hipMalloc(&d_maxValue, BlockSize*top_t*sizeof(float));
	hipMalloc(&h_top_t, top_t*sizeof(float));
	d_top_t = (float*)malloc(top_t*sizeof(float));
	//------------------------------------------------------------
	// Invoke kernel
	//------------------------------------------------------------
	GetMaxValue<<<gridSize,blockSize>>>(dev_data, \
									    data[0].num, data[1].num, \
										dim, \
										d_maxValue);
	hipDeviceSynchronize();
	int *potIdxBlock;
	hipMalloc(&potIdxBlock, BlockSize*sizeof(int));
	getPot<<<gridSize,1>>>(d_maxValue,potIdxBlock);
	hipDeviceSynchronize();
	mergeSort<<<1,1>>>(d_maxValue,d_top_t,potIdxBlock);
	//--------------------------------------------------
	// Copy result from device memory to host memory
	//--------------------------------------------------
	hipMemcpy(h_top_t, d_top_t, top_t*sizeof(float),hipMemcpyDeviceToHost);
	//----------------------------
	// write the top-t value to file
	//----------------------------
	FILE *fp_out = fopen(resultfile, "w");
	for (int i = 0; i < top_t; ++i){
		fprintf(fp_out,"%f\n",h_top_t[i]);
	}
	fclose(fp_out);
	//-----------------------------------
	// Free device memory and host memory
	//-----------------------------------
	for (int i = 0; i < numMat; ++i){
		hipFree(h_data[i].element);
		free(data[i].element); 
	}
	hipFree(dev_data);
	hipFree(d_maxValue);
	hipFree(d_top_t);
	hipFree(potIdxBlock);
	free(line);
	free(max_index);
	free(data);
	free(h_data);
	free(h_top_t);
	return 0;
}


__global__ void GetMaxValue(Matrix *dev_data,\
							size_t numXaxis, \
							size_t numYaxis, \
							int dim, \
							float *d_maxValue){
	// shared memory to save top 1024 value of each thread in this block
	__shared__ float cache[16*16*1024];
	__shared__ int potIdx[16*16];
	// the thread's potions
  	// for each thread in the block
	// the index of this thread is the block is
	// x = blockIdx.x * blockDim.x + threadIdx.x;
	// y = blockIdx.y * blockDim.y + threadIdx.y;
	const int2 thread_in_block = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
          		                            blockIdx.y * blockDim.y + threadIdx.y);
	// thread position with the matrix
	int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
          		                    blockIdx.y * blockDim.y + threadIdx.y);

  	// some intermediate variables
	float temp = 0.0;
	float *mulValue = (float*)malloc(dim*sizeof(float));
	// the result will be saved at cache_pos
  	const int cache_pos = (threadIdx.x + threadIdx.y * 16) * top_t;
 	// initialize the cache with the first value
 	// 0 may be bad for matrix are not guarantee to be positive
 	for(int d = 0; d < dim; ++d){
		temp +=  dev_data[0].element[thread_2D_pos.x*dim + d] *\
		           	dev_data[1].element[thread_2D_pos.y*dim + d]*\
		           	dev_data[2].element[d];
	} 	
	for(int i = 0; i < top_t; ++i){
		cache[cache_pos + i] = temp;
	}
  	// for there are too many works, thread will
  	// do a sub work xLoop * yLoop times 
  	size_t xLoop = (numXaxis + 16*64 - 1) / 16*64;
  	size_t yLoop = (numYaxis + 16*64 - 1) / 16*64;
  	// indicate the current work progress
  	struct cuSubIndex subIndex(xLoop,yLoop);

  	// do loop
  	while(!subIndex.isDone){
  		thread_2D_pos.x = thread_in_block.x + subIndex.x * 16*64;
  		thread_2D_pos.y = thread_in_block.y + subIndex.y * 16*64;
  		// out of range
  		if(thread_2D_pos.x >= numXaxis || thread_2D_pos.y >= numYaxis){
  			++subIndex;
  			continue;
  		}
		// compute X(i, j, k) for k in the range of Set C
		// i = thread_2D_pos.x
		// j = thread_2D_pos.y
		for(int d = 0; d < dim; ++d){
			mulValue[d] =  dev_data[0].element[thread_2D_pos.x*dim + d] *\
			           	dev_data[1].element[thread_2D_pos.y*dim + d];
		}
		// compute X(i, j, :) and put into the cache
		for(size_t k = 0; k < dev_data[2].num; ++k){
			temp = 0.0;
			for(int d = 0; d < dim; ++d){
				temp += dev_data[2].element[k*dim + d] * \
						mulValue[d];
			}
			if(temp > cache[cache_pos + top_t - 1]){
				doInsert(temp,(cache+cache_pos),top_t);
			}
		}
		++subIndex;
	}
	__syncthreads();

	//------------------------------------------------------
	// merge the cache to one vector containing the top-t value
	//------------------------------------------------------

	// there are 16 * 16 vectors,
	// for each vector called a beam
	// 16 * 16 is the same size of thread size per block;
	// each thread process on beam
	// find the top-s value in a beam are potential to 
	// be top-t of global
	// using all of top-s values in every beam 
	// we can get the global top-t values of this block
	int index = 0;
	bool doneSearch = false;
	for(int i = 0; i < top_t; ++i){
		// the order of cache[cache_pos + i] is at least i
		temp = cache[cache_pos + i];
		index = i;
		doneSearch = false;
		// compare this value to others
		// if encounter a value is other beam which bigger than it
		// increment index and go on, otherwise 
		// encountering a value no bigger than it
		// go to another beam to do comparison
		for(int m = 0; m < 16; ++m){
			for(int n = 0; n < 16; ++n){
				// escape self beam
				if(m == threadIdx.x && n == threadIdx.y){
					continue; 
				}
				// search in other beam
				for(int k = 0; k < top_t; ++k){
					// encounter a bigger value 
					if(temp < cache[(m + n * 16) * 1024 + k]){
						++index;
					}else{
						// this value is bigger than the left
						// of this beam, so no need to search deeper
						break;
					}
					// if this value is out of the top-t
					// we can finish searching
					if(index >= top_t){
						doneSearch = true;
						break;
					}
				}
				if(doneSearch) break;
			}
			if(doneSearch){
				break;
			}
		}
		// if beam[i] has no potential to be the top-t
		// then the number of elements have the potential
		// is i then record it;
		if(doneSearch){
			potIdx[threadIdx.x + threadIdx.y * 16] = i;
			break;
		}
  	}
  	
  	__syncthreads();
  	if(threadIdx.x == 0 && threadIdx.y == 0){
  		int count = 0;
 		for(int m = 0; m < 16; ++m){
			for(int n = 0; n < 16; ++n){
				count += potIdx[threadIdx.x + threadIdx.y * 16];
			}
		}
		float *potV = (float*)malloc(count*sizeof(float));
		int p = 0;
 		for(int m = 0; m < 16; ++m){
			for(int n = 0; n < 16; ++n){
				for(int i = 0 ; i < potIdx[threadIdx.x + threadIdx.y * 16];++i){
					potV[p] = cache[(m + n * 16) * 1024 + i];
					++p;
				}
			}
		}
		//sort potV
		for(int i = 1; i < count; i++){  
			if(potV[i] > potV[i-1]){
			    int j = i - 1;
			    float x = potV[i];
			    potV[i] = potV[i-1];
			    while(j >=0 && x > potV[j]){
			        potV[j + 1] = potV[j];
			        j--;
			    }
			    potV[j+1] = x;
			}  
		}
		for(int i = 0; i < top_t; ++i){
			d_maxValue[(blockIdx.x + blockIdx.y * 64)*1024 + i] = potV[i];
		}
		free(potV);
  	}
	free(mulValue);
}

__global__ void getPot(float*src, int *potIdxBlock){
	double temp;
	int index = 0;
	bool doneSearch = false;
	size_t pos = (blockIdx.x + blockIdx.y*64)*top_t;
	float *potV = (float*)malloc(64*64*sizeof(float));
	for(int i = 0; i < top_t; ++i){
		// the order of cache[cache_pos + i] is at least i
		temp = src[pos + i];
		index = i;
		doneSearch = false;
		// compare this value to others
		// if encounter a value is other beam which bigger than it
		// increment index and go on, otherwise 
		// encountering a value no bigger than it
		// go to another beam to do comparison
		for(int m = 0; m < blockDim.x; ++m){
			for(int n = 0; n < blockDim.y; ++n){
				// escape self beam
				if(m == blockDim.x && n == blockDim.y){
					continue; 
				}
				// search in other beam
				for(int k = 0; k < top_t; ++k){
					// encounter a bigger value 
					if(temp < src[pos + k]){
						++index;
					}else{
						// this value is bigger than the left
						// of this beam, so no need to search deeper
						break;
					}
					// if this value is out of the top-t
					// we can finish searching
					if(index >= top_t){
						doneSearch = true;
						break;
					}
				}
				if(doneSearch) break;
			}
			if(doneSearch){
				break;
			}
		}
		// if beam[i] has no potential to be the top-t
		// then the number of elements have the potential
		// is i then record it;
		if(doneSearch){
			potIdxBlock[(blockIdx.x + blockIdx.y*64)] = i;
			break;
		}
  	}
  	
}

__global__ void mergeSort(float*src,float*dst, int *potIdxBlock){
  	int count = 0;
 	for(int m = 0; m < 64; ++m){
		for(int n = 0; n < 64; ++n){
			count += potIdxBlock[m + n * 64];
		}
	}
	float *potV = (float*)malloc(count*sizeof(float));
	int p = 0;
 	for(int m = 0; m < 64; ++m){
		for(int n = 0; n < 64; ++n){
			for(int i = 0 ; i < potIdxBlock[m + n * 64];++i){
				potV[p] = src[(m + n * 64) * 1024 + i];
				++p;
			}
		}
	}
	//sort potV
	for(int i = 1; i < count; i++){  
		if(potV[i] > potV[i-1]){
		    int j = i - 1;
		    float x = potV[i];
		    potV[i] = potV[i-1];
		    while(j >= 0 && x > potV[j]){
		        potV[j + 1] = potV[j];
		        j--;
		    }
		    potV[j+1] = x;
		}  
	}
	for(int i = 0; i < top_t; ++i){
		dst[i] = potV[i];
	}
	free(potV);
}